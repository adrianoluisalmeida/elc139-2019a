#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>
#include <math.h>

__global__
void calculate(int width, int frames, unsigned char* pic)
{
  //Indice de segmento dentro do bloco
  int index = threadIdx.x;

  //This variable and contains the dimensions of the block.
  int offset = blockDim.x;

  for (int frame = index; frame < frames; frame += offset) {
    for (int row = 0; row < width; row++) {
      for (int col = 0; col < width; col++) {
        float fx = col - 1024/2;
        float fy = row - 1024/2;
        float d = sqrtf( fx * fx + fy * fy );
        unsigned char color = (unsigned char) (160.0f + 127.0f *
                                          cos(d/10.0f - frame/7.0f) /
                                          (d/50.0f + 1.0f));

        pic[frame * width * width + row * width + col] = (unsigned char) color;
      }
    }
  }
}

int main(int argc, char *argv[])
{

  // check command line
  if (argc != 3) {
    std::cout << "usage: " << std::endl;
     exit(-1);
  }
  int width = atoi(argv[1]);
  if (width < 100) {
    std::cout << "error: frame_width must be at least 100\n" << std::endl;
    exit(-1);
  }
  int frames = atoi(argv[2]);
  if (frames < 1) {
    std::cout << "error: num_frames must be at least 1\n" << std::endl; 
    exit(-1);
  }
  std::cout << "computing  " << std::cout << frames << " of " std::cout << " picture " << std::cout << width  << std::cout << " picture "  << std::endl;
  //printf("computing %d frames of %d by %d picture\n", frames, width, width);


  unsigned char* pic;

  //Allocating Unified Memory is as simple as replacing calls to malloc() or new with calls to hipMallocManaged(), an allocation function that   returns a pointer accessible from any processor (ptr in the following).
  hipMallocManaged(&pic, frames*width*width*sizeof(unsigned char));
    
  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // calculate threads for frames
  calculate<<<1, frames>>>(width, frames, pic);

  // hipDeviceSynchronize() will force the program to ensure the stream(s)'s kernels/memcpys are complete before continuing
  hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  std::cout<< "compute time: " << runtime << std::endl;

  // verify result by writing frames to BMP files
  //if ((width <= 256) && (frames <= 100)) {
  //  for (int frame = 0; frame < frames; frame++) {
  //    char name[32];
  //  //  sprintf(name, "wave%d.bmp", frame + 1000);
  //   // writeBMP(width, width, &pic[frame * width * width], name);
  //  }
  //}
  hipFree(pic);
  return 0;
}
